#include "hip/hip_runtime.h"
/**
 * Calculate the kinetic energies of each degree of freedom.
 */

extern "C" __global__ void computeDrudeTGNHKineticEnergies(mixed4* __restrict__ velm, 
        const int* __restrict__ normalParticles, const int2* __restrict__ pairParticles,
        mixed* __restrict__ normalKE, mixed* __restrict__ realKE, mixed* __restrict__ drudeKE) {
 
    // Add kinetic energy of ordinary particles.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            normalKE[i] = (velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w;
        }
        else
            normalKE[i] = 0;
    }
    
    // Add kinetic energy of Drude particle pairs.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        mixed4 velocity1 = velm[particles.x];
        mixed4 velocity2 = velm[particles.y];
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed invReducedMass = (mass1+mass2)*velocity1.w*velocity2.w;
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velocity1*mass1fract+velocity2*mass2fract;
        mixed4 relVel = velocity2-velocity1;

        realKE[i] = (cmVel.x*cmVel.x + cmVel.y*cmVel.y + cmVel.z*cmVel.z)*(mass1+mass2);
        drudeKE[i] = (relVel.x*relVel.x + relVel.y*relVel.y + relVel.z*relVel.z)*RECIP(invReducedMass);
    }
}

extern "C" __global__ void sumDrudeKineticEnergies(mixed* __restrict__ normalKE, mixed* __restrict__ realKE,
        mixed* __restrict__ drudeKE, mixed* __restrict__ kineticEnergies) {

    __shared__ mixed normalKESum[WORK_GROUP_SIZE];
    __shared__ mixed realKESum[WORK_GROUP_SIZE];
    __shared__ mixed drudeKESum[WORK_GROUP_SIZE];

    unsigned int tid = threadIdx.x;

    normalKESum[tid] = 0;
    realKESum[tid] = 0;
    drudeKESum[tid] = 0;

    for (unsigned int index = tid; index < NUM_NORMAL_PARTICLES; index += blockDim.x) {
        normalKESum[tid] += normalKE[index];
    }


    for (unsigned int index = tid; index < NUM_PAIRS; index += blockDim.x) {
        realKESum[tid] += realKE[index];
        drudeKESum[tid] += drudeKE[index];
    }

    __syncthreads();
    for (int i = WORK_GROUP_SIZE/2; i>0; i>>=1) {
        if (tid < i) {
            normalKESum[tid] += normalKESum[tid + i];
            realKESum[tid] += realKESum[tid + i];
            drudeKESum[tid] += drudeKESum[tid + i];
        }
        __syncthreads();
    }

    if (tid == 0) {
        kineticEnergies[0] = normalKESum[0] + realKESum[0];
        kineticEnergies[1] = drudeKESum[0];
    }
}

/**
 * Calculate the center of mass velocities of each residues and get relative velocities of each particles
 */

extern "C" __global__ void calcCOMVelocities(const mixed4* __restrict__ velm,
        const int2* __restrict__ particlesInResidues, mixed4* __restrict__ comVelm, bool useCOMTempGroup) {

    // Get COM velocities
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_RESIDUES; i += blockDim.x*gridDim.x) {
        comVelm[i] = make_mixed4(0,0,0,0);
        if (useCOMTempGroup) {
        mixed comMass = 0.0;
        for (int j = 0; j < particlesInResidues[i].x; j++) {
            int index = particlesInResidues[i].y + j;
            mixed4 velocity = velm[index];
            if (velocity.w != 0) {
                mixed mass = RECIP(velocity.w);
                comVelm[i].x += velocity.x * mass;
                comVelm[i].y += velocity.y * mass;
                comVelm[i].z += velocity.z * mass;
                comMass += mass;
            }
        }
        comVelm[i].w = RECIP(comMass);
        comVelm[i].x *= comVelm[i].w;
        comVelm[i].y *= comVelm[i].w;
        comVelm[i].z *= comVelm[i].w;
        }
        else {
            comVelm[i].w = 1.0;
        }
        //if (i==0)
        //    printf("residue %d has %d particles and starts at %d and vel %f,%f,%f and mass is %f \n",i,particlesInResidues[i].x,particlesInResidues[i].y, comVelm[i].x,comVelm[i].y,comVelm[i].z, RECIP(comVelm[i].w));
    }

}

/**
 * Calculate the center of mass velocities of each residues and get relative velocities of each particles
 */

extern "C" __global__ void normalizeVelocities(const mixed4* __restrict__ velm, const int* __restrict__ particleResId,
        const mixed4* __restrict__ comVelm, mixed4* __restrict__ normVelm) {

    // Get Normalized velocities
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        normVelm[i] = make_mixed4(0,0,0,0);
        int resid = particleResId[i];
        normVelm[i].x = velm[i].x - comVelm[resid].x;
        normVelm[i].y = velm[i].y - comVelm[resid].y;
        normVelm[i].z = velm[i].z - comVelm[resid].z;
        normVelm[i].w = velm[i].w;
        //if (i==0)
        //    printf("Particle : %d, Norm Velocity : %f, velocity : %f, comVel : %f, mass : %f  \n", i,normVelm[i].x, velm[i].x, comVelm[resid].x, RECIP(normVelm[i].w));
    }
}
/**
 * Calculate the kinetic energies of each degree of freedom.
 */

extern "C" __global__ void computeNormalizedKineticEnergies(const mixed4* __restrict__ comVelm,
        const mixed4* __restrict__ normVelm, const int* __restrict__ particleTempGroup,
        const int* __restrict__ normalParticles, const int2* __restrict__ pairParticles,
        double* __restrict__ kineticEnergyBuffer) {

    unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
    for (int i=0; i < NUM_TEMP_GROUPS+2; i++)
        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+i] = 0;

    //double comKE = 0;
    //double realKE[NUM_TEMP_GROUPS] = {0};
    //double drudeKE = 0;

    // Add kinetic energy of ordinary particles.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_RESIDUES; i += blockDim.x*gridDim.x) {
        mixed4 velocity = comVelm[i];
        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+NUM_TEMP_GROUPS] += (velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w;
        //comKE += (velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w;
        //printf("i %d, comKE %f, tid %u, vx %f, vy %f, vz %f, vw %f, calc %f \n",i,comKE,tid,velocity.x,velocity.y,velocity.z,velocity.w,(velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w);
        
    }

    // Add kinetic energy of ordinary particles.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = normVelm[index];
        if (velocity.w != 0) {
            kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+particleTempGroup[index]] += (velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w;
        //    realKE[particleTempGroup[index]] += (velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z)/velocity.w;
        }
    }

    // Add kinetic energy of Drude particle pairs.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        mixed4 velocity1 = normVelm[particles.x];
        mixed4 velocity2 = normVelm[particles.y];
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed invReducedMass = (mass1+mass2)*velocity1.w*velocity2.w;
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velocity1*mass1fract+velocity2*mass2fract;
        mixed4 relVel = velocity2-velocity1;

        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+particleTempGroup[particles.x]] += (cmVel.x*cmVel.x + cmVel.y*cmVel.y + cmVel.z*cmVel.z)*(mass1+mass2);
        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+NUM_TEMP_GROUPS+1] += (relVel.x*relVel.x + relVel.y*relVel.y + relVel.z*relVel.z)*RECIP(invReducedMass);
        //realKE[particleTempGroup[particles.x]] += (cmVel.x*cmVel.x + cmVel.y*cmVel.y + cmVel.z*cmVel.z)*(mass1+mass2);
        //drudeKE += (relVel.x*relVel.x + relVel.y*relVel.y + relVel.z*relVel.z)*RECIP(invReducedMass);
    }
    __syncthreads();

//    for (int i=0; i < NUM_TEMP_GROUPS; i++)
//        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+i] = realKE[i];
//    kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+NUM_TEMP_GROUPS] = comKE;
//    kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+NUM_TEMP_GROUPS+1] = drudeKE;
//    __syncthreads();
//    if (tid==0) {
//        printf("comKE %f, realKE %f, drude %f tid %u \n",comKE,realKE[0],drudeKE,tid);
//        printf("blockdim %d griddim %d num_temp_group %d buffersize %d comKE %f, realKE %f, drude %f tid %u \n",blockDim.x,gridDim.x,NUM_TEMP_GROUPS+2,int(*(&kineticEnergyBuffer+1)-kineticEnergyBuffer),comKE,realKE[0],drudeKE,tid);
//    }
}

extern "C" __global__ void sumNormalizedKineticEnergies(double* __restrict__ kineticEnergyBuffer, double* __restrict__ kineticEnergies, int bufferSize) {
    // Sum the threads in this group.
    __shared__ double temp[WORK_GROUP_SIZE*(NUM_TEMP_GROUPS+2)];
    unsigned int tid = threadIdx.x;

    for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i++)
        temp[WORK_GROUP_SIZE*i+tid] = 0;
    __syncthreads();

    for (unsigned int index = tid*(NUM_TEMP_GROUPS+2); index < bufferSize; index += blockDim.x*(NUM_TEMP_GROUPS+2)) {
        for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i++)
            temp[WORK_GROUP_SIZE*i+tid] += kineticEnergyBuffer[index + i];
    }
    __syncthreads();
    if (tid < 32) {
        for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i+=1) temp[WORK_GROUP_SIZE*i+tid] += temp[WORK_GROUP_SIZE*i+tid+32];
        __syncthreads();
        if (tid < 16) {
            for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i+=1) temp[WORK_GROUP_SIZE*i+tid] += temp[WORK_GROUP_SIZE*i+tid+16];
        }
        __syncthreads();
        if (tid < 8) {
            for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i+=1) temp[WORK_GROUP_SIZE*i+tid] += temp[WORK_GROUP_SIZE*i+tid+8];
        }
        __syncthreads();
        if (tid < 4) {
            for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i+=1) temp[WORK_GROUP_SIZE*i+tid] += temp[WORK_GROUP_SIZE*i+tid+4];
        }
        __syncthreads();
        if (tid < 2) {
            for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i+=1) temp[WORK_GROUP_SIZE*i+tid] += temp[WORK_GROUP_SIZE*i+tid+2];
        }
        __syncthreads();
    }
    __syncthreads();
    if (tid == 0) {
        for (unsigned int i=0; i < NUM_TEMP_GROUPS+2; i++) {
            kineticEnergies[i] = temp[WORK_GROUP_SIZE*i]+temp[WORK_GROUP_SIZE*i+1];
        }
    }
}


/**
 * Perform the velocity update of TGNH Chain integration.
 */

extern "C" __global__ void integrateDrudeTGNHChain(mixed4* __restrict__ velm, const mixed4* __restrict__ normVelm,
        const int* __restrict__ normalParticles, const int2* __restrict__ pairParticles, const int* __restrict__ particleTempGroup, const mixed* __restrict__ vscaleFactors) {

    mixed vscaleCOM = vscaleFactors[NUM_TEMP_GROUPS];
    mixed vscaleDrude = vscaleFactors[NUM_TEMP_GROUPS+1];
    // Update normal particles.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = velm[index];
        mixed4 velRel = normVelm[index];
        mixed vscale = vscaleFactors[particleTempGroup[index]];
        if (velocity.w != 0) {
            velocity.x = vscale*velRel.x + vscaleCOM*(velocity.x-velRel.x);
            velocity.y = vscale*velRel.y + vscaleCOM*(velocity.y-velRel.y);
            velocity.z = vscale*velRel.z + vscaleCOM*(velocity.z-velRel.z);
            velm[index] = velocity;
        }
    }
    
    // Update Drude particle pairs.
    
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        mixed vscaleCM = vscaleFactors[particleTempGroup[particles.x]];
        mixed4 velocity1 = velm[particles.x];
        mixed4 velocity2 = velm[particles.y];
        mixed4 velRel1 = normVelm[particles.x];
        mixed4 velRel2 = normVelm[particles.y];
        mixed4 velCOM1 = velocity1 - velRel1;
        mixed4 velCOM2 = velocity2 - velRel2;
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velRel1*mass1fract+velRel2*mass2fract;
        mixed4 relVel = velRel2-velRel1;
        cmVel.x = vscaleCM*cmVel.x;
        cmVel.y = vscaleCM*cmVel.y;
        cmVel.z = vscaleCM*cmVel.z;
        relVel.x = vscaleDrude*relVel.x;
        relVel.y = vscaleDrude*relVel.y;
        relVel.z = vscaleDrude*relVel.z;
        velocity1.x = cmVel.x-relVel.x*mass2fract + vscaleCOM*velCOM1.x;
        velocity1.y = cmVel.y-relVel.y*mass2fract + vscaleCOM*velCOM1.y;
        velocity1.z = cmVel.z-relVel.z*mass2fract + vscaleCOM*velCOM1.z;
        velocity2.x = cmVel.x+relVel.x*mass1fract + vscaleCOM*velCOM2.x;
        velocity2.y = cmVel.y+relVel.y*mass1fract + vscaleCOM*velCOM2.y;
        velocity2.z = cmVel.z+relVel.z*mass1fract + vscaleCOM*velCOM2.z;
        velm[particles.x] = velocity1;
        velm[particles.y] = velocity2;
    }
}

/**
 * Perform the velocity update of TGNH Chain integration.
 */

extern "C" __global__ void integrateDrudeTGNHVelocities(mixed4* __restrict__ velm, const long long* __restrict__ force, mixed4* __restrict__ posDelta,
        const int* __restrict__ normalParticles, const int2* __restrict__ pairParticles, const mixed2* __restrict__ dt, const mixed fscale,
        const mixed fscaleDrude, bool updatePosDelta) {
    mixed stepSize = dt[0].y;
    
    // Update normal particles.

    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            velocity.x = velocity.x + fscale*velocity.w*force[index];
            velocity.y = velocity.y + fscale*velocity.w*force[index+PADDED_NUM_ATOMS];
            velocity.z = velocity.z + fscale*velocity.w*force[index+PADDED_NUM_ATOMS*2];
            velm[index] = velocity;
            if (updatePosDelta) {
                posDelta[index] = make_mixed4(stepSize*velocity.x, stepSize*velocity.y, stepSize*velocity.z, 0);
            }
        }
    }
    
    // Update Drude particle pairs.
    
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        mixed4 velocity1 = velm[particles.x];
        mixed4 velocity2 = velm[particles.y];
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed invReducedMass = (mass1+mass2)*velocity1.w*velocity2.w;
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velocity1*mass1fract+velocity2*mass2fract;
        mixed4 relVel = velocity2-velocity1;
        mixed3 force1 = make_mixed3(force[particles.x], force[particles.x+PADDED_NUM_ATOMS], force[particles.x+PADDED_NUM_ATOMS*2]);
        mixed3 force2 = make_mixed3(force[particles.y], force[particles.y+PADDED_NUM_ATOMS], force[particles.y+PADDED_NUM_ATOMS*2]);
        mixed3 cmForce = force1+force2;
        mixed3 relForce = force2*mass1fract - force1*mass2fract;
        cmVel.x = cmVel.x + fscale*invTotalMass*cmForce.x;
        cmVel.y = cmVel.y + fscale*invTotalMass*cmForce.y;
        cmVel.z = cmVel.z + fscale*invTotalMass*cmForce.z;
        relVel.x = relVel.x + fscaleDrude*invReducedMass*relForce.x;
        relVel.y = relVel.y + fscaleDrude*invReducedMass*relForce.y;
        relVel.z = relVel.z + fscaleDrude*invReducedMass*relForce.z;
        velocity1.x = cmVel.x-relVel.x*mass2fract;
        velocity1.y = cmVel.y-relVel.y*mass2fract;
        velocity1.z = cmVel.z-relVel.z*mass2fract;
        velocity2.x = cmVel.x+relVel.x*mass1fract;
        velocity2.y = cmVel.y+relVel.y*mass1fract;
        velocity2.z = cmVel.z+relVel.z*mass1fract;
        velm[particles.x] = velocity1;
        velm[particles.y] = velocity2;
        if (updatePosDelta) {
            posDelta[particles.x] = make_mixed4(stepSize*velocity1.x, stepSize*velocity1.y, stepSize*velocity1.z, 0);
            posDelta[particles.y] = make_mixed4(stepSize*velocity2.x, stepSize*velocity2.y, stepSize*velocity2.z, 0);
        }
    }
}

/**
 * Perform the velocity update of TGNH Chain integration.
 */

extern "C" __global__ void integrateDrudeTGNHVelocitiesAndPositions(mixed4* __restrict__ velm, const long long* __restrict__ force, mixed4* __restrict__ posDelta,
        const int* __restrict__ normalParticles, const int2* __restrict__ pairParticles, const mixed2* __restrict__ dt, const mixed vscale, const mixed fscale,
        const mixed vscaleDrude, const mixed fscaleDrude, bool updatePosDelta) {
    mixed stepSize = dt[0].y;
    
    // Update normal particles.

    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            velocity.x = vscale*velocity.x + fscale*velocity.w*force[index];
            velocity.y = vscale*velocity.y + fscale*velocity.w*force[index+PADDED_NUM_ATOMS];
            velocity.z = vscale*velocity.z + fscale*velocity.w*force[index+PADDED_NUM_ATOMS*2];
            velm[index] = velocity;
            if (updatePosDelta) {
                posDelta[index] = make_mixed4(stepSize*velocity.x, stepSize*velocity.y, stepSize*velocity.z, 0);
            }
        }
    }
    
    // Update Drude particle pairs.
    
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        mixed4 velocity1 = velm[particles.x];
        mixed4 velocity2 = velm[particles.y];
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed invReducedMass = (mass1+mass2)*velocity1.w*velocity2.w;
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velocity1*mass1fract+velocity2*mass2fract;
        mixed4 relVel = velocity2-velocity1;
        mixed3 force1 = make_mixed3(force[particles.x], force[particles.x+PADDED_NUM_ATOMS], force[particles.x+PADDED_NUM_ATOMS*2]);
        mixed3 force2 = make_mixed3(force[particles.y], force[particles.y+PADDED_NUM_ATOMS], force[particles.y+PADDED_NUM_ATOMS*2]);
        mixed3 cmForce = force1+force2;
        mixed3 relForce = force2*mass1fract - force1*mass2fract;
        cmVel.x = vscale*cmVel.x + fscale*invTotalMass*cmForce.x;
        cmVel.y = vscale*cmVel.y + fscale*invTotalMass*cmForce.y;
        cmVel.z = vscale*cmVel.z + fscale*invTotalMass*cmForce.z;
        relVel.x = vscaleDrude*relVel.x + fscaleDrude*invReducedMass*relForce.x;
        relVel.y = vscaleDrude*relVel.y + fscaleDrude*invReducedMass*relForce.y;
        relVel.z = vscaleDrude*relVel.z + fscaleDrude*invReducedMass*relForce.z;
        velocity1.x = cmVel.x-relVel.x*mass2fract;
        velocity1.y = cmVel.y-relVel.y*mass2fract;
        velocity1.z = cmVel.z-relVel.z*mass2fract;
        velocity2.x = cmVel.x+relVel.x*mass1fract;
        velocity2.y = cmVel.y+relVel.y*mass1fract;
        velocity2.z = cmVel.z+relVel.z*mass1fract;
        velm[particles.x] = velocity1;
        velm[particles.y] = velocity2;
        if (updatePosDelta) {
            posDelta[particles.x] = make_mixed4(stepSize*velocity1.x, stepSize*velocity1.y, stepSize*velocity1.z, 0);
            posDelta[particles.y] = make_mixed4(stepSize*velocity2.x, stepSize*velocity2.y, stepSize*velocity2.z, 0);
        }
    }
}

/**
 * Perform the position update of TGNH integration.
 */

extern "C" __global__ void integrateDrudeTGNHPositions(real4* __restrict__ posq, real4* __restrict__ posqCorrection, const mixed4* __restrict__ posDelta, mixed4* __restrict__ velm, const mixed2* __restrict__ dt) {
    double invStepSize = 1.0/dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < NUM_ATOMS) {
        mixed4 vel = velm[index];
        if (vel.w != 0) {
#ifdef USE_MIXED_PRECISION
 
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            mixed4 delta = posDelta[index];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
            vel.x = (mixed) (invStepSize*delta.x);
            vel.y = (mixed) (invStepSize*delta.y);
            vel.z = (mixed) (invStepSize*delta.z);
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
            velm[index] = vel;
        }
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Apply hard wall constraints
 */
extern "C" __global__ void applyHardWallConstraints(real4* __restrict__ posq, real4* __restrict__ posqCorrection, mixed4* __restrict__ velm,
        const int2* __restrict__ pairParticles, const mixed2* __restrict__ dt, mixed maxDrudeDistance, mixed hardwallscaleDrude) {
    mixed stepSize = dt[0].y;
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
#ifdef USE_MIXED_PRECISION
        real4 posReal1 = posq[particles.x];
        real4 posReal2 = posq[particles.y];
        real4 posCorr1 = posqCorrection[particles.x];
        real4 posCorr2 = posqCorrection[particles.y];
        mixed4 pos1 = make_mixed4(posReal1.x+(mixed)posCorr1.x, posReal1.y+(mixed)posCorr1.y, posReal1.z+(mixed)posCorr1.z, posReal1.w);
        mixed4 pos2 = make_mixed4(posReal2.x+(mixed)posCorr2.x, posReal2.y+(mixed)posCorr2.y, posReal2.z+(mixed)posCorr2.z, posReal2.w);
#else
        mixed4 pos1 = posq[particles.x];
        mixed4 pos2 = posq[particles.y];
#endif
        mixed4 delta = pos1-pos2;
        mixed r = SQRT(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
        mixed rInv = RECIP(r);
        if (rInv*maxDrudeDistance < 1) {
            // The constraint has been violated, so make the inter-particle distance "bounce"
            // off the hard wall.

            mixed4 bondDir = delta*rInv;
            mixed4 vel1 = velm[particles.x];
            mixed4 vel2 = velm[particles.y];
            mixed mass1 = RECIP(vel1.w);
            mixed mass2 = RECIP(vel2.w);
            mixed deltaR = r-maxDrudeDistance;
            mixed deltaT = stepSize;
            mixed dotvr1 = vel1.x*bondDir.x + vel1.y*bondDir.y + vel1.z*bondDir.z;
            mixed4 vb1 = bondDir*dotvr1;
            mixed4 vp1 = vel1-vb1;
            if (vel2.w == 0) {
                // The parent particle is massless, so move only the Drude particle.

                if (dotvr1 != 0)
                    deltaT = deltaR/fabs(dotvr1);
                if (deltaT > stepSize)
                    deltaT = stepSize;
                dotvr1 = -dotvr1*hardwallscaleDrude/(fabs(dotvr1)*SQRT(mass1));
                mixed dr = -deltaR + deltaT*dotvr1;
                pos1.x += bondDir.x*dr;
                pos1.y += bondDir.y*dr;
                pos1.z += bondDir.z*dr;
#ifdef USE_MIXED_PRECISION
                posq[particles.x] = make_real4((real) pos1.x, (real) pos1.y, (real) pos1.z, (real) pos1.w);
                posqCorrection[particles.x] = make_real4(pos1.x-(real) pos1.x, pos1.y-(real) pos1.y, pos1.z-(real) pos1.z, 0);
#else
                posq[particles.x] = pos1;
#endif
                vel1.x = vp1.x + bondDir.x*dotvr1;
                vel1.y = vp1.y + bondDir.y*dotvr1;
                vel1.z = vp1.z + bondDir.z*dotvr1;
                velm[particles.x] = vel1;
            }
            else {
                // Move both particles.

                mixed invTotalMass = RECIP(mass1+mass2);
                mixed dotvr2 = vel2.x*bondDir.x + vel2.y*bondDir.y + vel2.z*bondDir.z;
                mixed4 vb2 = bondDir*dotvr2;
                mixed4 vp2 = vel2-vb2;
                mixed vbCMass = (mass1*dotvr1 + mass2*dotvr2)*invTotalMass;
                dotvr1 -= vbCMass;
                dotvr2 -= vbCMass;
                if (dotvr1 != dotvr2)
                    deltaT = deltaR/fabs(dotvr1-dotvr2);
                if (deltaT > stepSize)
                    deltaT = stepSize;
                mixed vBond = hardwallscaleDrude/SQRT(mass1);
                dotvr1 = -dotvr1*vBond*mass2*invTotalMass/fabs(dotvr1);
                dotvr2 = -dotvr2*vBond*mass1*invTotalMass/fabs(dotvr2);
                mixed dr1 = -deltaR*mass2*invTotalMass + deltaT*dotvr1;
                mixed dr2 = deltaR*mass1*invTotalMass + deltaT*dotvr2;
                dotvr1 += vbCMass;
                dotvr2 += vbCMass;
                pos1.x += bondDir.x*dr1;
                pos1.y += bondDir.y*dr1;
                pos1.z += bondDir.z*dr1;
                pos2.x += bondDir.x*dr2;
                pos2.y += bondDir.y*dr2;
                pos2.z += bondDir.z*dr2;
#ifdef USE_MIXED_PRECISION
                posq[particles.x] = make_real4((real) pos1.x, (real) pos1.y, (real) pos1.z, (real) pos1.w);
                posq[particles.y] = make_real4((real) pos2.x, (real) pos2.y, (real) pos2.z, (real) pos2.w);
                posqCorrection[particles.x] = make_real4(pos1.x-(real) pos1.x, pos1.y-(real) pos1.y, pos1.z-(real) pos1.z, 0);
                posqCorrection[particles.y] = make_real4(pos2.x-(real) pos2.x, pos2.y-(real) pos2.y, pos2.z-(real) pos2.z, 0);
#else
                posq[particles.x] = pos1;
                posq[particles.y] = pos2;
#endif
                vel1.x = vp1.x + bondDir.x*dotvr1;
                vel1.y = vp1.y + bondDir.y*dotvr1;
                vel1.z = vp1.z + bondDir.z*dotvr1;
                vel2.x = vp2.x + bondDir.x*dotvr2;
                vel2.y = vp2.y + bondDir.y*dotvr2;
                vel2.z = vp2.z + bondDir.z*dotvr2;
                velm[particles.x] = vel1;
                velm[particles.y] = vel2;
            }
        }
    }
}
